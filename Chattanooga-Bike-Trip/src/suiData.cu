#include "suiData.h"

BikeTrip* parse_data(const char* fileName, BikeTrip* bTrips, size_t* nRides) {
    std::ifstream ifs(fileName, std::ifstream::in);
    std::string line, point, xpos, ypos;
    std::vector<BikeTrip> trips;

    if (!ifs.is_open()) {
        std::cerr << "File did not open correctly" << std::endl;
        exit(EXIT_FAILURE);
    }

    getline(ifs, line); // Column Names
    int i = 0;
    while (getline(ifs, line)) {
        bool push = true;
        std::stringstream ss(line);
        BikeTrip trip;
        getline(ss, line, ','); // Member Type
        getline(ss, line, ','); // Bike ID
        trip.BikeID = line;
        getline(ss, line, ','); // Start Time
        getline(ss, line, ','); // Start Station Name
        getline(ss, line, ','); // Start Station ID
        getline(ss, line, ','); // Start Location
        
        // START LOCATION
        std::istringstream iss(line);
        iss >> point >> xpos >> ypos;
        try {
            trip.startingX = std::stod(xpos.c_str() + 1);
            trip.startingY = std::stod(ypos.c_str() + 0);
        }
        catch (std::invalid_argument) {
            push = false;
            continue;
        }

        xpos.clear();
        ypos.clear();

        getline(ss, line, ','); // End Time
        getline(ss, line, ','); // End Station Name
        getline(ss, line, ','); // End Station ID
        getline(ss, line, ','); // End Location

        // END LOCATION
        iss.clear();
        iss.str(line);
        iss >> point >> xpos >> ypos;
        
        try {
            trip.endX = std::stod(xpos.c_str() + 1);
            trip.endY = std::stod(ypos.c_str() + 0);
        }
        catch (std::invalid_argument) {
            push = false;
            continue;
        }

        // RIDE TIME
        getline(ss, line, ','); // Trip Duration Min
        trip.rideTime = stod(line);

        if(push) trips.push_back(trip);
        xpos.clear();
        ypos.clear();
        i++;
    }

    // Vector to Array
    *nRides = trips.size();
    bTrips = new BikeTrip[*nRides];
    std::copy(trips.begin(), trips.end(), bTrips);
    ifs.close();
    return bTrips;
}

void write_to_file(BikeTrip* trips, TripStats* stats, int size) {
    std::ofstream ofs;
    ofs.open("output.csv", std::ofstream::out);
    ofs << "BikeID,DistanceInYards,YardsPerMin" << std::endl;
    int idx = 0;
    while (idx < size) {
        if (stats[idx].distanceInYard) {
            ofs << trips[idx].BikeID << "," << stats[idx].distanceInYard << "," << stats[idx].yardPerMin << std::endl;
        }
        idx++;
    }
}