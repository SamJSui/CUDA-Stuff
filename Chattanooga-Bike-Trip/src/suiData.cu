#include "suiData.h"

std::vector<BikeTrip> parse_data(const char* fileName) {
    std::ifstream ifs(fileName, std::ifstream::in);
    std::string line, point, xpos, ypos;
    std::vector<BikeTrip> trips;
    
    if (!ifs.is_open()) {
        std::cerr << "File did not open correctly" << std::endl;
        exit(EXIT_FAILURE);
    }

    getline(ifs, line); // Column Names

    while (getline(ifs, line)) {
        std::stringstream ss(line);
        BikeTrip trip;
        getline(ss, line, ','); // Member Type
        getline(ss, line, ','); // Bike ID
        getline(ss, line, ','); // Start Time
        getline(ss, line, ','); // Start Station Name
        getline(ss, line, ','); // Start Station ID
        getline(ss, line, ','); // Start Location
        
        // START LOCATION
        std::istringstream iss(line);
        iss >> point >> xpos >> ypos;
        trip.startingX = atof(xpos.c_str() + 1);
        trip.startingY = atof(ypos.c_str() + 0);

        getline(ss, line, ','); // End Time
        getline(ss, line, ','); // End Station Name
        getline(ss, line, ','); // End Station ID
        getline(ss, line, ','); // End Location
        
        // END LOCATION
        iss.clear();
        iss.str(line);
        iss >> point >> xpos >> ypos;
        trip.endX = atof(xpos.c_str() + 1);
        trip.endY = atof(ypos.c_str() + 0);
        
        // RIDE TIME
        getline(ss, line, ','); // Trip Duration Min
        trip.rideTime = stof(line);

        trips.push_back(trip);
    }
    return trips;
}