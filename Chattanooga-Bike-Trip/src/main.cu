#include "suiData.h"
#include "kernel.cuh"

int main (int argc, char** argv) {
    const char* fileName;
    if (argc < 2) {
        std::cerr << "Too few arguments!" << std::endl;
        return 1;
    } else {
        fileName = argv[1];
    }
    
    size_t nRides = 0;
    BikeTrip* trips = parse_data(fileName, trips, &nRides), *d_trips;
    const int trip_bytes = nRides * sizeof(BikeTrip);
    const int stat_bytes = nRides * sizeof(TripStats);
    printf("nRides: %zu\n", nRides);
    TripStats* stats = new TripStats[nRides], *d_stats;

    hipError_t malloc_trip = hipMalloc((void**) &d_trips, trip_bytes); // Allocates memory size of buf onto Device
    if (malloc_trip != hipSuccess){
        printf("%s", hipGetErrorString( malloc_trip ));
    }else{
        printf("%s\n", "TRIPS: CUDAMALLOC COMPLETE");
    }
    
    hipError_t malloc_stats = hipMalloc((void**) &d_stats, stat_bytes); // Allocates memory size of buf onto Device
    if (malloc_stats != hipSuccess){
        printf("%s", hipGetErrorString( malloc_stats ));
    }else{
        printf("%s\n", "STATS: CUDAMALLOC STATS COMPLETE");
    }

    hipError_t tripMemcpyHtoD = hipMemcpy(d_trips, trips, trip_bytes, hipMemcpyHostToDevice); // Copies memory from Host to Device
    if (tripMemcpyHtoD != hipSuccess){
        printf("%s\n", hipGetErrorString( tripMemcpyHtoD ));
    }else{
        printf("%s\n", "TRIPS: MEMCPY FROM HOST TO DEVICE COMPLETE");
    }

    dim3 nThreads(512, 1, 1); 
    dim3 nBlocks(nRides/nThreads.x, 1, 1);
    calculate_distance<<<nBlocks, nThreads>>>(d_trips, d_stats, nRides);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));

    hipError_t cudaSync = hipDeviceSynchronize();
    if (cudaSync != hipSuccess){
        printf("%s\n", hipGetErrorString( cudaSync ));
    }else{
        printf("%s\n", "CUDA SYNC SUCCESS");
    }

    hipError_t statMemcpyDtoH = hipMemcpy(stats, d_stats, stat_bytes, hipMemcpyDeviceToHost); // Copies memory from Host to Device
    if (statMemcpyDtoH != hipSuccess){
        printf("%s\n", hipGetErrorString( statMemcpyDtoH ));
    }else{
        printf("%s\n", "STATS: MEMCPY FROM DEVICE TO HOST COMPLETE");
    }

    write_to_file(trips, stats, nRides);

    hipFree(d_trips);
    hipFree(d_stats);
    delete[] stats;
    delete[] trips;
    return 0;
}