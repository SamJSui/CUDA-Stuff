#include <iostream> 
#include <fstream>
#include "suiData.h"

int main (int argc, char** argv) {
    const char* fileName;
    if (argc < 2) {
        std::cerr << "Too few arguments!" << std::endl;
        return 1;
    } else {
        fileName = argv[1];
    }
    parse_data(fileName);
    return 0;
}