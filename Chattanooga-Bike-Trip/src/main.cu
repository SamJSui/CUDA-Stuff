#include "suiData.h"

int main (int argc, char** argv) {
    const char* fileName;
    if (argc < 2) {
        std::cerr << "Too few arguments!" << std::endl;
        return 1;
    } else {
        fileName = argv[1];
    }

    std::vector<BikeTrip> trips = parse_data(fileName);
    std::cout << trips.size() << std::endl;
    
    return 0;
}