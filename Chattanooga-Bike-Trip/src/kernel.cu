#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__
void calculate_distance(BikeTrip* trips, TripStats* stats, int size) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < size) {
        double dx = trips[idx].endX - trips[idx].startingX;
        double dy = trips[idx].endY - trips[idx].startingY;
        dx = dx * (1000/9) * 3280.4 / 3; // Longitude/Latitude to km
        dy = dy * (1000/9) * 3280.4 / 3;
        stats[idx].distanceInYard = sqrt(pow(dx,2)+pow(dy,2));
        stats[idx].yardPerMin = stats[idx].distanceInYard / trips[idx].rideTime;
    }
}